#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include "timer.c"

#include "kmeans.h"

#if defined GPU
// From the NVIDIA CUDA programming guide. No idea how it works
__device__ double atomicAdd(double *address, double val)
{
    double old = *address, assumed;
    do {
        assumed = old;
        old = __longlong_as_double(atomicCAS((unsigned long long int*)address,
                                             __double_as_longlong(assumed),
                                             __double_as_longlong(val + assumed)));
    } while (assumed != old);
    return old;
}

__global__ void ResetCentroidForEachCluster(Cluster *clusters)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;
    // Set clusters[].noOfPoints to 0. Only for K centroids
    if (pt < K) {
        clusters[pt].noOfPoints = 0;
    }
}

__global__ void ComputeClusters(Point *points, Cluster *clusters, Point *tempPoints)
{
    int pt = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    double max;
    int inCluster;

    if (pt >= N)
        return;

    // Save the old centroid and clear the x and y components of
    // each point. I'm going to use first K of these to store
    // the sum of co-ordinates of points in this cluster.
    // clusterId field is used to save old centroid for each point
    // so that we know when to stop iterating.
    tempPoints[pt].clusterId = points[pt].clusterId;
    tempPoints[pt].loc[X_AXIS] = 0.0;
    tempPoints[pt].loc[Y_AXIS] = 0.0;

    // Compute the nearest centroid.
    max = GetDistanceGPU(points[pt], clusters[0].pt);
    inCluster = 0;
    for (i = 0; i < K; i++) {
        if (GetDistanceGPU(points[pt], clusters[i].pt) < max) {
            inCluster = i;
            max = GetDistanceGPU(points[pt], clusters[i].pt);
        }
    }
    atomicAdd(&clusters[inCluster].noOfPoints, 1);
    // Bottle neck.
    atomicAdd(&tempPoints[inCluster].loc[X_AXIS], points[pt].loc[X_AXIS]);
    atomicAdd(&tempPoints[inCluster].loc[Y_AXIS], points[pt].loc[Y_AXIS]);

    points[pt].clusterId = inCluster;
}

__global__ void ComputeCentroids(Cluster *clusters, Point *tempPoints)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;

    // Now calculate the new centroids.
    if (pt < K) {
        clusters[pt].pt.loc[X_AXIS] = tempPoints[pt].loc[X_AXIS]/clusters[pt].noOfPoints;
        clusters[pt].pt.loc[Y_AXIS] = tempPoints[pt].loc[Y_AXIS]/clusters[pt].noOfPoints;
    }
}

__global__ void RepeatNeeded(Point *points, Point *tempPoints, unsigned int *key)
{
    int pt = blockIdx.x * blockDim.x + threadIdx.x;

    if (pt < N) {
        if (points[pt].clusterId != tempPoints[pt].clusterId) {
                *key = 1;
        }
    }

}

void DoKmeansGPU (Point *points, Cluster *clusters)
{
    Point *dPoints, *dTempPoints;
    Cluster *dClusters;
    unsigned int *repeat, repeatHost;

    hipMalloc ((void **)&dPoints, sizeof(Point)*N);
    hipMalloc ((void **)&dClusters, sizeof(Cluster)*K);
    hipMalloc ((void **)&dTempPoints, sizeof(Point)*N);
    hipMalloc ((void **)&repeat, sizeof(unsigned int));

    hipMemcpy(dPoints, points, sizeof(Point)*N, hipMemcpyHostToDevice);
    hipMemcpy(dClusters, clusters, sizeof(Cluster)*K, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid (N / threadsPerBlock.x);

    do {
        ResetCentroidForEachCluster <<< blocksPerGrid, threadsPerBlock >>> (dClusters);
        ComputeClusters <<< blocksPerGrid, threadsPerBlock >>> (dPoints, dClusters, dTempPoints);
        ComputeCentroids <<< blocksPerGrid, threadsPerBlock >>> (dClusters, dTempPoints);

        hipMemset(repeat, 0, sizeof(unsigned int));

        RepeatNeeded <<<blocksPerGrid, threadsPerBlock >>> (dPoints, dTempPoints, repeat);

        hipMemcpy(&repeatHost, repeat, sizeof(unsigned int), hipMemcpyDeviceToHost);
    } while (repeatHost);


    hipMemcpy(points, dPoints, sizeof(Point)*N, hipMemcpyDeviceToHost);
    hipMemcpy(clusters, dClusters, sizeof(Cluster)*K, hipMemcpyDeviceToHost);

    hipFree(dPoints);
    hipFree(dClusters);
    hipFree(dTempPoints);
    hipFree(repeat);
}
#endif // definedGPU

void DoKmeansCPU (Point *points, Cluster *clusters)
{

    double max;
    int i, j, inCluster;
    bool changed;
    Point *tempPoints;

    // One for each cluster (and point). For cluster, use x and y, for point, clusterId.
    tempPoints  = (Point *) malloc (sizeof(Point)*N);

    do {

        memset(tempPoints, 0, sizeof(Point)*N);
        for (i = 0; i < K; i++) {
            clusters[i].noOfPoints = 0;
        }
        // Save the old clusterId for each point. Reusing tempPoints
        for (i = 0; i < N; i++) {
            tempPoints[i].clusterId = points[i].clusterId;
        }
        // For each point, find the nearest centroid.
        for (i = 0; i < N; i++) {
            max = GetDistance(points[i], clusters[0].pt);
            inCluster = 0;
            for (j = 0; j < K; j++) {
                if (GetDistance(points[i], clusters[j].pt) < max) {
                    inCluster = j;
                    // TODO: Store these distances, instead of re-computing
                    // (I don't mean from above call, I mean totally for the program).
                    max = GetDistance(points[i], clusters[j].pt);
                }
            }
            clusters[inCluster].noOfPoints++;
            tempPoints[inCluster].loc[X_AXIS] += points[i].loc[X_AXIS];
            tempPoints[inCluster].loc[Y_AXIS] += points[i].loc[Y_AXIS];
            points[i].clusterId = inCluster;
        }

        // Compute new centroid for each cluster
        for (i = 0; i < K; i++) {
            // Assuming that each cluster has atleast one point in it.
            assert(clusters[i].noOfPoints != 0);
            clusters[i].pt.loc[X_AXIS] = tempPoints[i].loc[X_AXIS]/clusters[i].noOfPoints;
            clusters[i].pt.loc[Y_AXIS] = tempPoints[i].loc[Y_AXIS]/clusters[i].noOfPoints;
        }

        // Check if anything has changed
        changed = false;
        for (i = 0; i < N; i++) {
            if (points[i].clusterId != tempPoints[i].clusterId) {
                changed = true;
                break;
            }
        }
    } while (changed);

}

int main (int argc, char *argv[])
{

    Point *pointsCPU;
    Cluster *clustersCPU;
    int i, j;

#ifdef GPU
    Point *pointsGPU;
    Cluster *clustersGPU;
#endif

    srandom(time(NULL));

    pointsCPU = (Point *) malloc(sizeof(Point)*N);
    clustersCPU = (Cluster *) malloc(sizeof(Cluster)*K);

    // Get the points randomly
    for (i = 0; i < N; i++) {
        for (int d=0; d < DIMENSIONS; d++) {
          pointsCPU[i].loc[d] = (random()/1021322);
        }
        pointsCPU[i].clusterId = -1;
    }

    // Initialize clusters
    for (i = 0; i < K; i++) {
        clustersCPU[i].pt.clusterId = i;
        clustersCPU[i].noOfPoints = 0;
        j = random()%N;
        if (pointsCPU[j].clusterId != -1) {
            i--; continue;
            // Potential infinite loop
        }
        pointsCPU[j].clusterId = i;
        clustersCPU[i].pt.loc[X_AXIS] = pointsCPU[j].loc[X_AXIS];
        clustersCPU[i].pt.loc[Y_AXIS] = pointsCPU[j].loc[Y_AXIS];
    }

#ifdef DEBUG
    printf ("Initial points:\n");
    for (i = 0; i < N; i++) {
        printf ("x=%.2f, y=%.2f, clusterId=%d\n", pointsCPU[i].loc[X_AXIS], pointsCPU[i].loc[Y_AXIS], pointsCPU[i].clusterId);
    }
    printf ("Initial clusters:\n");
    for (i = 0; i < K; i++) {
        printf("clusterId=%d, noOfPoints=%d, centroidX=%.2f, centroidY=%.2f\n", clustersCPU[i].pt.clusterId,
               clustersCPU[i].noOfPoints, clustersCPU[i].pt.loc[X_AXIS], clustersCPU[i].pt.loc[Y_AXIS]);
    }
#endif // DEBUG

#ifdef GPU
    pointsGPU = (Point *) malloc (sizeof(Point)*N);
    clustersGPU = (Cluster *) malloc (sizeof(Cluster)*K);

    memcpy(pointsGPU, pointsCPU, sizeof(Point)*N);
    memcpy(clustersGPU, clustersCPU, sizeof(Cluster)*K);

    tstart();
    DoKmeansGPU(pointsGPU, clustersGPU);
    tend();
    printf("%f seconds on GPU.\n", tval());
#endif

    // Data for other versions are copied from here. So don't want it
    // to change before copying.
    tstart();
    DoKmeansCPU(pointsCPU, clustersCPU);
    tend();
    printf("%f seconds on CPU.\n", tval());


#ifdef PRETTY_PRINT
#ifdef GPU
    // Showing GPU dumps
    FILE *fp; char buf[20];
    system ("rm /tmp/*plot");
    for (i = 0; i < N; i++) {
        sprintf(buf, "/tmp/%d.plot", pointsGPU[i].clusterId);
        fp = fopen (buf, "a");
        if (fp) {
            fprintf (fp, "%.2f %.2f #%d GPU\n", pointsGPU[i].loc[X_AXIS], pointsGPU[i].loc[Y_AXIS], pointsGPU[i].clusterId);
            fclose(fp);
        }
    }
#else
    // Showing CPU dumps
    FILE *fp; char buf[20];
    system ("rm /tmp/*plot");
    for (i = 0; i < N; i++) {
        sprintf(buf, "/tmp/%d.plot", pointsCPU[i].clusterId);
        fp = fopen (buf, "a");
        if (fp) {
            fprintf (fp, "%.2f %.2f #%d CPU\n", pointsCPU[i].loc[X_AXIS], pointsCPU[i].loc[Y_AXIS], pointsCPU[i].clusterId);
            fclose(fp);
        }
    }
#endif // #if defined (GPU)
#endif // PRETTY_PRINT

    return 0;
}
